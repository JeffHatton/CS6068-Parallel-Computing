#include "hip/hip_runtime.h"
// Homework 1
// Color to Greyscale Conversion

//A common way to represent color images is known as RGBA - the color
//is specified by how much Red, Grean and Blue is in it.
//The 'A' stands for Alpha and is used for transparency, it will be
//ignored in this homework.

//Each channel Red, Blue, Green and Alpha is represented by one byte.
//Since we are using one byte for each color there are 256 different
//possible values for each color.  This means we use 4 bytes per pixel.

//Greyscale images are represented by a single intensity value per pixel
//which is one byte in size.

//To convert an image from color to grayscale one simple method is to
//set the intensity to the average of the RGB channels.  But we will
//use a more sophisticated method that takes into account how the eye 
//perceives color and weights the channels unequally.

//The eye responds most strongly to green followed by red and then blue.
//The NTSC (National Television System Committee) recommends the following
//formula for color to greyscale conversion:

//I = .299f * R + .587f * G + .114f * B

//Notice the trailing f's on the numbers which indicate that they are 
//single precision floating point constants and not double precision
//constants.

//You should fill in the kernel as well as set the block and grid sizes
//so that the entire image is processed.

#include "utils.h"

__global__
void rgba_to_greyscale(const uchar4* const rgbaImage,
                       unsigned char* const greyImage,
                       int numRows, int numCols)
{
	// Get x and y coordinates
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;

	// Calculate the pixel id
	int id = idx + idy * numCols;

	// If the x or y coordinate is bigger than the number of columns or rows return as it 
	// is out of the image
	if (idx > numCols || idy > numRows) return;

	// pull the pixel out of the array
	uchar4 pixel = rgbaImage[id];

	// Scale the pixel into grey scale
	greyImage[id] = pixel.x * .299f + pixel.y * .587f + pixel.z *.114f;
}

void your_rgba_to_greyscale(const uchar4 * const h_rgbaImage, uchar4 * const d_rgbaImage,
                            unsigned char* const d_greyImage, size_t numRows, size_t numCols)
{
	int blockSi = 32;
	int gridSizeCol =  std::ceil(numCols / 32.0f);
	int gridSizeRows = std::ceil(numRows / 32.0f);

	const dim3 blockSize(32, 32, 1);  //TODO
	const dim3 gridSize(gridSizeCol, gridSizeRows, 1);  //TODO
	rgba_to_greyscale<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImage, numRows, numCols);
  
	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

}
